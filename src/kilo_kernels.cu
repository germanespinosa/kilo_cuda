#include "hip/hip_runtime.h"
#include "kilo_kernels.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

Robot      *cuda_robots;
Position   *cuda_next_positions;
Rectangle  *cuda_light_shapes;

Robot local_robots[ROBOTS];


__global__ void compute_step(Robot *robots, Position *next_position);
__global__ void collision_and_comms(Robot *robots,Position *next_position);
__global__ void update_state(Robot *robots,Position *next_position);
__global__ void initialize_robot_data_kernel(Robot *robots, Position *positions);
__global__ void commpute_step_kernel(Robot *robots,Step *step);
__global__ void compute_light(Robot *robots, Rectangle *cuda_light_shapes);

static dim3 lingrid(1,1);
static dim3 cuadgrid(ROBOTS,1);
static dim3 block(ROBOTS,1);
static dim3 shapesgrid;

void initialize_shapes(Rectangle *rectangles, int shapecount)
{
    // Upload shapes to GPU memory (light/shapes are static; occurs once at initialization)
    hipMalloc((void**)&cuda_light_shapes, sizeof(Rectangle) * shapecount);
	dim3 grid(shapecount,1);
	shapesgrid = grid;
 	hipMemcpy(cuda_light_shapes, rectangles, sizeof(Rectangle) * shapecount, hipMemcpyHostToDevice);
}

void initialize_robots(Position *positions)
{
    // Upload initial robots/positions to GPU memory
    hipMalloc((void**)&cuda_robots, sizeof(Robot) * ROBOTS);
	hipMalloc((void**)&cuda_next_positions, sizeof(Position) * ROBOTS);
	hipMemcpy(cuda_next_positions, positions, sizeof(Position) * ROBOTS, hipMemcpyHostToDevice);
	initialize_robot_data_kernel <<< lingrid, block >>> ( cuda_robots, cuda_next_positions );
}

void simulation_step()
{
    // Compute next positions/communications
	compute_step <<< lingrid, block >>> ( cuda_robots, cuda_next_positions );
    // Compute light sensor values from shapes
	compute_light <<< shapesgrid, block >>> ( cuda_robots, cuda_light_shapes );
    // Check if communcations/movements valid
	collision_and_comms <<< cuadgrid, block >>> ( cuda_robots, cuda_next_positions );
    // Update next state from validity checks
	update_state <<< lingrid, block >>> ( cuda_robots, cuda_next_positions );
	// Download robot data to CPU
	hipMemcpy(local_robots, cuda_robots, sizeof(Robot) * ROBOTS, hipMemcpyDeviceToHost);
	// Repopulate robots state
	for (int rid=0;rid<ROBOTS;rid++)
	{
		// execute controller loop
	}
	hipMemcpy(cuda_robots, local_robots, sizeof(Robot) * ROBOTS, hipMemcpyHostToDevice);
	// upload state changes
	
}

Robot *download_robot_data()
{
 	hipMemcpy(local_robots, cuda_robots, sizeof(Robot) * ROBOTS, hipMemcpyDeviceToHost);
    return local_robots;
}

__global__ void compute_light(Robot *robots, Rectangle *cuda_light_shapes)
{
    // Calculate light sensor values from rectangles
    unsigned int sid = blockIdx.x;
	unsigned int rid = threadIdx.x;
    // Check if robot is in shape
    // If it is, set light to 1000
    // TODO: How to check/reset light to 0 at each time step (essentially want 1 output ["any"/"or"] from all combined)
    // TODO: How to deal with border area (gray light; maybe have to convert the way this is checked?)
}

__global__ void compute_step(Robot *robots, Position *next_position)
{
    unsigned int rid = threadIdx.x;
	//compute the movement needed	
	Step step;
	if (robots[rid].left_motor == 0) robots[rid].left_motor_active = false;
	if (robots[rid].right_motor == 0) robots[rid].right_motor_active = false;

	float turn_error  = HRAND * robots[rid].movement.turn_error - robots[rid].movement.turn_error / 2;
	float speed_error = HRAND * robots[rid].movement.speed_error - robots[rid].movement.speed_error / 2;
	
	step.turn = robots[rid].left_motor_active ? (robots[rid].right_motor_active ? robots[rid].movement.turn_forward: robots[rid].movement.turn_left) : (robots[rid].right_motor_active ? robots[rid].movement.turn_right: 0);
	step.speed = robots[rid].left_motor_active ? (robots[rid].right_motor_active ? robots[rid].movement.speed_forward: robots[rid].movement.speed_left) : (robots[rid].right_motor_active ? robots[rid].movement.speed_right: 0);
	
	step.turn += step.turn ? turn_error : 0 ;
	step.speed += step.speed ? speed_error : 0 ;
	
	
	//compute the next position	
    robots[rid].position.theta+=step.turn;
	Position temp_p;
    temp_p.theta = robots[rid].position.theta;
    temp_p.x = robots[rid].position.x + cos(robots[rid].position.theta) * step.speed;
    temp_p.y = robots[rid].position.y + sin(robots[rid].position.theta) * step.speed;
    if (INBOUNDS(temp_p))
    {
		next_position[rid]=temp_p;
	}
	else
    {
		next_position[rid]=robots[rid].position;
	}
}

__global__ void collision_and_comms(Robot *robots, Position *next_position)
{
    unsigned int nid = blockIdx.x;
    unsigned int rid = threadIdx.x;
	float d = DIST(robots[rid].position, next_position[nid]);
	float range_error = robots[rid].comm.range_error*HRAND - robots[rid].comm.range_error / 2;
	float range=robots[rid].comm.range + range_error;
	
	if (robots[nid].tx_flag && HRAND>robots[rid].comm.range_error && d<range)
	{
		robots[nid].rx_flag=true;
		for (int i=0;i<MESSAGE_SIZE;i++)
		{
			robots[nid].message_rx[i] = robots[rid].message_rx[i];
		}
	}
	
	if (d<ROBOT_RADIUS)
	{
		next_position[nid]=robots[nid].position;
	}
}

__global__ void update_state(Robot *robots,Position *next_position)
{
    unsigned int rid = threadIdx.x;
	robots[rid].position=next_position[rid];
	robots[rid].tx_flag=false;
}

__global__ void initialize_robot_data_kernel(Robot *robots, Position *positions)
{
    unsigned int rid = threadIdx.x;
	// initialize random seeds
	// all robots have the same soft random sequence
	hiprand_init(0, 0, 0, &robots[rid].sstate);
	// all robots have different hard random sequence
	hiprand_init(rid, 0, 0, &robots[rid].hstate);

	// intitialize position
	robots[rid].position.x += positions[rid].x;
	robots[rid].position.y += positions[rid].y;
    robots[rid].position.theta += positions[rid].theta;
	
	// initialize movement parameters 
	// turn
	robots[rid].movement.turn_left = - (MIN_TURN + (HRAND * (MAX_TURN - MIN_TURN)));
	robots[rid].movement.turn_right = (MIN_TURN + (HRAND * (MAX_TURN - MIN_TURN)));
	robots[rid].movement.turn_forward = (MIN_TURN + (HRAND * (MAX_TURN - MIN_TURN)));
	robots[rid].movement.turn_forward -= robots[rid].movement.turn_forward / 2;
	robots[rid].movement.turn_error = MAX_TURN_ERROR * HRAND;
	// speed
	robots[rid].movement.speed_left = (MIN_SPEED + (HRAND * (MAX_SPEED - MIN_SPEED)));
	robots[rid].movement.speed_right = (MIN_SPEED + (HRAND * (MAX_SPEED - MIN_SPEED)));
	robots[rid].movement.speed_forward = (MIN_SPEED + (HRAND * (MAX_SPEED - MIN_SPEED)));
	robots[rid].movement.speed_error = MAX_SPEED_ERROR * HRAND;
	
	// initialize comm parameters
	robots[rid].comm.comm_error = MAX_COMM_ERROR * HRAND; //probability of message not being transmitted 
	robots[rid].comm.range = (MIN_COMM_RANGE + (HRAND * (MAX_COMM_RANGE - MIN_COMM_RANGE)));
	robots[rid].comm.range_error = MAX_COMM_RANGE_ERROR * HRAND;
	
	//comms
	robots[rid].tx_flag=false;
	robots[rid].rx_flag=false;

	//motors
	robots[rid].left_motor=0;
	robots[rid].left_motor_active=false;
	robots[rid].right_motor=0;
	robots[rid].right_motor_active=false;
}