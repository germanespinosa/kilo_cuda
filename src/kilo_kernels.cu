#include "hip/hip_runtime.h"
#include "kilo_kernels.h"
#include "kilolib.h"
#include "kilocode.cpp"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

static Robot         *cuda_robots;
static Position      *cuda_next_positions;
static Rectangle     *cuda_light_shapes;
static hiprandState_t *cuda_rand_states;

static Robot local_robots[ROBOTS];

//kernels prototypes

__global__ void compute_step_kernel(Robot *robots, Position *next_position, hiprandState_t *rand_states);
__global__ void collision_and_comms_kernel(Robot *robots,Position *next_position, hiprandState_t *rand_states);
__global__ void update_state_kernel(Robot *robots,Position *next_positio, hiprandState_t *rand_statesn);
__global__ void initialize_robot_data_kernel(Robot *robots, Position *positions, hiprandState_t *rand_states);
__global__ void compute_step_kernel(Robot *robots,Step *step, hiprandState_t *rand_states);
__global__ void compute_light_kernel(Robot *robots, Rectangle *cuda_light_shapes, hiprandState_t *rand_states);

static dim3 lingrid(LINGRID,1);
static dim3 block(TILELIMIT,1);
static dim3 shapesgrid;

void release_cuda_memory()
{
    hipFree(cuda_robots);
    hipFree(cuda_next_positions);
    hipFree(cuda_robots);
}

void initialize_shapes(Rectangle *rectangles, int shapecount)// add upload shapes.
{
    hipMalloc((void**)&cuda_light_shapes, sizeof(Rectangle) * shapecount);
	dim3 grid(LINGRID,shapecount);
	shapesgrid = grid;
 	hipMemcpy(cuda_light_shapes, rectangles, sizeof(Rectangle) * shapecount, hipMemcpyHostToDevice);
}
// floats(x1,y1)(x2,y2) int(r,g,b)

void initialize_robots(Position *positions) //
{
    hipMalloc((void**)&cuda_robots, sizeof(Robot) * ROBOTS);
	hipMalloc((void**)&cuda_rand_states, sizeof(hiprandState_t) * ROBOTS * 2);
	hipMalloc((void**)&cuda_next_positions, sizeof(Position) * ROBOTS);
	hipMemcpy(cuda_next_positions, positions, sizeof(Position) * ROBOTS, hipMemcpyHostToDevice);
	initialize_robot_data_kernel <<< lingrid, block >>> ( cuda_robots, cuda_next_positions, cuda_rand_states);
}

void simulation_step()
{
	Kilo_Impl kilobot;
	compute_step_kernel <<< lingrid, block >>> ( cuda_robots, cuda_next_positions, cuda_rand_states);
	//compute_light <<< shapesgrid, block >>> ( cuda_robots, cuda_light_shapes );
	collision_and_comms_kernel <<< lingrid, block >>> ( cuda_robots, cuda_next_positions, cuda_rand_states);
	//collision_and_comms << < lingrid, block >> > (cuda_robots, cuda_next_positions);
	update_state_kernel <<< lingrid, block >>> ( cuda_robots, cuda_next_positions, cuda_rand_states);
	// download data
	hipMemcpy(local_robots, cuda_robots, sizeof(Robot) * ROBOTS, hipMemcpyDeviceToHost);
	// repopulate robots state
	for (int rid=0;rid<ROBOTS;rid++)
	{
		// execute controller loop
		kilobot.run_controller(local_robots + rid );
		//julias function (local_robots (*ROBOTS))
	}
	hipMemcpy(cuda_robots, local_robots, sizeof(Robot) * ROBOTS, hipMemcpyHostToDevice);
	// upload state changes
	
}

Robot *download_robot_data()
{
 	hipMemcpy(local_robots, cuda_robots, sizeof(Robot) * ROBOTS, hipMemcpyDeviceToHost);
    return local_robots;
}

void release_cuda_memory()
{
	hipFree(cuda_robots);
	hipFree(cuda_next_positions);
	hipFree(cuda_robots);
	hipFree(cuda_rand_states);
	hipDeviceReset();
}

__global__ void compute_light_kernel(Robot *robots, Rectangle *light_shapes, hiprandState_t *rand_states)
{
//    unsigned int sid = SHAPEID;
//	unsigned int rid = ROBOTID;
	//if robot rid is in shape sid 
	// robot[rid] light = value. 
}

__global__ void compute_step_kernel(Robot *robots, Position *next_position, hiprandState_t *rand_states)
{
    unsigned int rid = ROBOTID;
	//compute the movement needed	
	Step step;
	if (robots[rid].left_motor == 0) robots[rid].left_motor_active = false;
	if (robots[rid].right_motor == 0) robots[rid].right_motor_active = false;

	float turn_error  = HRAND * robots[rid].movement.turn_error - robots[rid].movement.turn_error / 2;
	float speed_error = HRAND * robots[rid].movement.speed_error - robots[rid].movement.speed_error / 2;
	
	step.turn = robots[rid].left_motor_active ? (robots[rid].right_motor_active ? robots[rid].movement.turn_forward: robots[rid].movement.turn_left) : (robots[rid].right_motor_active ? robots[rid].movement.turn_right: 0);
	step.speed = robots[rid].left_motor_active ? (robots[rid].right_motor_active ? robots[rid].movement.speed_forward: robots[rid].movement.speed_left) : (robots[rid].right_motor_active ? robots[rid].movement.speed_right: 0);;
	
	step.turn += step.turn ? turn_error : 0 ;
	step.speed += step.speed ? speed_error : 0 ;
	
	
	//compute the next position	
    robots[rid].position.theta+=step.turn;
	Position temp_p;
    temp_p.theta = robots[rid].position.theta;
    temp_p.x = robots[rid].position.x + cos(robots[rid].position.theta) * step.speed;
    temp_p.y = robots[rid].position.y + sin(robots[rid].position.theta) * step.speed;
    if (INBOUNDS(temp_p))
    {
		next_position[rid]=temp_p;
	}
	else
    {
		next_position[rid]=robots[rid].position;
	}
}

__global__ void collision_and_comms_kernel(Robot *robots,Position *next_position, hiprandState_t *rand_states)
{
	unsigned int rid = ROBOTID;
//	unsigned int nid = blockIdx.x;
	bool colide = false;
	for (int nid = 0; nid < rid && !colide; nid++)
	{
		float d = DIST(next_position[rid],robots[nid].position);
		if (d < ROBOT_RADIUS)
		{
			next_position[rid] = robots[rid].position;
			colide = true;
		}
	}
	if (robots[rid].tx_flag)
	{
		float range_error = robots[rid].comm.range_error*HRAND - robots[rid].comm.range_error / 2;
		float range = robots[rid].comm.range + range_error;
		for (int nid = 0; nid < ROBOTS ; nid++)
		{
			float d = DIST(next_position[rid], robots[nid].position);
			if (d < range)
			{
				if (HRAND > robots[rid].comm.comm_error)
				{
					robots[nid].rx_flag = true;
					robots[nid].rx_distance = d + range_error;
					for (int i = 0; i < MESSAGE_SIZE; i++)
					{
						robots[nid].message_rx[i] = robots[rid].message_rx[i];
					}
				}
			}
		}
	}
}

__global__ void update_state_kernel(Robot *robots,Position *next_position, hiprandState_t *rand_states)
{
    unsigned int rid = ROBOTID;
	robots[rid].position=next_position[rid];
	robots[rid].tx_flag=false;
}

__global__ void initialize_robot_data_kernel(Robot *robots, Position *positions, hiprandState_t *rand_states)
{
    unsigned int rid = ROBOTID;
	// initialize random seeds
	// all robots have the same soft random sequence
	hiprand_init(0, 0, 0, rand_states + rid);

	// all robots have different hard random sequence
	hiprand_init(rid, 0, 0, rand_states + ROBOTS + rid );

	// intitialize position
	robots[rid].position.theta += positions[rid].theta;
	robots[rid].position.x += positions[rid].x;
	robots[rid].position.y += positions[rid].y;

    // TODO: Include battery initialization (Gaussian distributed)
	
	// initialize movement parameters 
	// turn
	robots[rid].movement.turn_left = - (MIN_TURN + (HRAND * (MAX_TURN - MIN_TURN)));
	robots[rid].movement.turn_right = (MIN_TURN + (HRAND * (MAX_TURN - MIN_TURN)));
	robots[rid].movement.turn_forward = (MIN_TURN + (HRAND * (MAX_TURN - MIN_TURN)));
	robots[rid].movement.turn_forward -= robots[rid].movement.turn_forward / 2;
	robots[rid].movement.turn_error = MAX_TURN_ERROR * HRAND;
	// speed
	robots[rid].movement.speed_left = (MIN_SPEED + (HRAND * (MAX_SPEED - MIN_SPEED)));
	robots[rid].movement.speed_right = (MIN_SPEED + (HRAND * (MAX_SPEED - MIN_SPEED)));
	robots[rid].movement.speed_forward = (MIN_SPEED + (HRAND * (MAX_SPEED - MIN_SPEED)));
	robots[rid].movement.speed_error = MAX_SPEED_ERROR * HRAND;
	
	// initialize comm parameters
	robots[rid].comm.comm_error = MAX_COMM_ERROR * HRAND; //probability of message not being transmitted 
	robots[rid].comm.range = (MIN_COMM_RANGE + (HRAND * (MAX_COMM_RANGE - MIN_COMM_RANGE)));
	robots[rid].comm.range_error = MAX_COMM_RANGE_ERROR * HRAND;
	
	//comms
	robots[rid].tx_flag=false;
	robots[rid].rx_flag=false;

	//motors
	robots[rid].left_motor=0;
	robots[rid].left_motor_active=false;
	robots[rid].right_motor=0;
	robots[rid].right_motor_active=false;
}